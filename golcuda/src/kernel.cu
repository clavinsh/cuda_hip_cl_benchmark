#include "hip/hip_runtime.h"
// Game Of Life implementācija CUDA vidē

#include "benchmarkLogger.h"
#include <assert.h>
#include <cassert>
#include <chrono>
#include <cstring>
#include <cuda/std/cstdint> // analogs C/C++ <cstdint>, bet nodrošina fiksētus datu tipu lielumus uz device
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <utility>
#include <vector>

// macro priekš katra cuda API izsaukuma rezultāta pārbaudes
// ņemts no
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CUDA_CHECK(ans)                                                                                                \
	{                                                                                                                  \
		gpuAssert((ans), __FILE__, __LINE__);                                                                          \
	}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

// izveido flat grid masīvu, automātiski nosakot width, height
// met ārā kļūdas ja nav atbilstošu simbolu (1, 0) vai ja kāda rindiņa nesatur tādu pašu simbolu skaitu kā pirmā
std::vector<unsigned char> loadGridFromFile(const std::string &fileName, size_t &width, size_t &height)
{
	// ejam uz faila beigām uzreiz ar 'ate', lai noteiktu faila izmēru, pēc tam iesim uz sākumu
	std::ifstream file(fileName, std::ios::ate | std::ios::binary);

	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t fileSize = file.tellg();
	file.seekg(0, std::ios::beg);

	std::vector<char> buffer(fileSize);
	file.read(buffer.data(), fileSize);
	file.close();

	std::vector<unsigned char> grid;
	grid.reserve(fileSize);

	size_t lineStartPos = 0;

	width = 0;
	height = 0;

	for (size_t i = 0; i <= buffer.size(); ++i)
	{
		if (i == buffer.size() || buffer[i] == '\n')
		{
			size_t lineLen = i - lineStartPos;

			if (lineLen == 0)
			{
				lineStartPos = i + 1;
				continue;
			}

			if (width == 0)
			{
				width = lineLen;
			}
			else if (lineLen != width)
			{
				throw std::runtime_error("Invalid line length at line idx: " + std::to_string(height));
			}

			for (size_t j = 0; j < width; ++j)
			{
				unsigned char val = static_cast<unsigned char>(buffer[lineStartPos + j] - '0');
				grid.push_back(val);
			}

			height++;
			lineStartPos = i + 1;
		}
	}

	return grid;
}

void writeGridToFile(std::vector<unsigned char> &grid, size_t width, size_t height, std::string fileName)
{
	std::ofstream file(fileName, std::ios::out | std::ios::binary);
	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t totalSize = (width + 1) * height; // +1, jo rindas beigās ir \n

	std::vector<char> buffer(totalSize);

	for (size_t h = 0; h < height; h++)
	{
		size_t lineStart = h * (width + 1);
		size_t gridRowStart = h * width;

		for (size_t w = 0; w < width; w++)
		{
			buffer[lineStart + w] = '0' + grid[gridRowStart + w];
		}

		buffer[lineStart + width] = '\n';
	}

	file.write(buffer.data(), totalSize);

	file.close();
}

__constant__ size_t d_width;
__constant__ size_t d_height;

// Simplified CUDA kernel more closely matching the OpenCL version
__global__ void golMultiStepKernel(const unsigned char *input, unsigned char *output, unsigned char *temp,
								   size_t stepsToProcess)
{
	// Thread indices
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if thread is within grid bounds (early exit)
	if (x >= d_width || y >= d_height)
		return;

	const size_t flatIdx = y * d_width + x;

	// Copy input to temp
	temp[flatIdx] = input[flatIdx];

	// Ensure all threads have written to temp
	__syncthreads();

	for (size_t step = 0; step < stepsToProcess; step++)
	{
		// Ping-pong buffers (same as OpenCL version)
		const unsigned char *curr = (step % 2 == 0) ? temp : output;
		unsigned char *next = (step % 2 == 0) ? output : temp;

		// Count neighbors directly from global memory (like OpenCL version)
		int neighbors = 0;

		// Top row
		if (y > 0)
		{
			// Top-left
			if (x > 0)
				neighbors += curr[(y - 1) * d_width + (x - 1)];

			// Top-center
			neighbors += curr[(y - 1) * d_width + x];

			// Top-right
			if (x < d_width - 1)
				neighbors += curr[(y - 1) * d_width + (x + 1)];
		}

		// Middle row (excluding center)
		if (x > 0)
			neighbors += curr[y * d_width + (x - 1)]; // Middle-left

		if (x < d_width - 1)
			neighbors += curr[y * d_width + (x + 1)]; // Middle-right

		// Bottom row
		if (y < d_height - 1)
		{
			// Bottom-left
			if (x > 0)
				neighbors += curr[(y + 1) * d_width + (x - 1)];

			// Bottom-center
			neighbors += curr[(y + 1) * d_width + x];

			// Bottom-right
			if (x < d_width - 1)
				neighbors += curr[(y + 1) * d_width + (x + 1)];
		}

		// Apply Game of Life rules (same logic as OpenCL version)
		unsigned char cell = 0;
		if (curr[flatIdx] == 1)
		{
			if (neighbors == 2 || neighbors == 3)
				cell = 1;
		}
		else
		{
			if (neighbors == 3)
				cell = 1;
		}

		next[flatIdx] = cell;

		// Ensure all threads complete before next iteration
		__syncthreads();
	}

	// If odd number of steps, ensure output has the final state
	if (stepsToProcess % 2 == 1)
	{
		output[flatIdx] = temp[flatIdx];
	}
}

// Main function to run Game of Life simulation with CUDA
void GameOfLifeStep(std::vector<unsigned char> &grid, std::vector<unsigned char> &outputGrid, size_t width,
					size_t height, size_t steps, BenchmarkLogger &logger)
{

	const size_t stepsPerKernel = 1 << 10; // 1024 steps per kernel invocation (same as OpenCL)
	size_t gridSize = width * height;
	outputGrid.resize(gridSize);

	auto start = std::chrono::steady_clock::now();

	// Allocate pinned memory for faster host-device transfers
	unsigned char *hostPinnedInput = nullptr;
	unsigned char *hostPinnedOutput = nullptr;
	CUDA_CHECK(hipHostMalloc(&hostPinnedInput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipHostMalloc(&hostPinnedOutput, gridSize * sizeof(unsigned char)));

	// Copy grid to pinned memory
	std::memcpy(hostPinnedInput, grid.data(), gridSize * sizeof(unsigned char));

	// Copy width and height to constant memory
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(size_t)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(size_t)));

	// Allocate device memory
	unsigned char *deviceInput = nullptr;
	unsigned char *deviceOutput = nullptr;
	unsigned char *deviceTemp = nullptr;
	CUDA_CHECK(hipMalloc(&deviceInput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipMalloc(&deviceOutput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipMalloc(&deviceTemp, gridSize * sizeof(unsigned char)));

	auto end = std::chrono::steady_clock::now();
	logger.chronoLog("buffer creation time", start, end);

	// Transfer data to device
	start = std::chrono::steady_clock::now();
	hipEvent_t transferEvent, startEvent, endEvent;
	CUDA_CHECK(hipEventCreate(&transferEvent));
	CUDA_CHECK(hipEventCreate(&startEvent));
	CUDA_CHECK(hipEventCreate(&endEvent));

	CUDA_CHECK(hipEventRecord(startEvent));
	CUDA_CHECK(hipMemcpy(deviceInput, hostPinnedInput, gridSize * sizeof(unsigned char), hipMemcpyHostToDevice));
	CUDA_CHECK(hipEventRecord(transferEvent));
	CUDA_CHECK(hipEventSynchronize(transferEvent));

	float transferTime = 0;
	CUDA_CHECK(hipEventElapsedTime(&transferTime, startEvent, transferEvent));
	logger.log("host-to-device transfer time", transferTime);

	end = std::chrono::steady_clock::now();
	logger.chronoLog("total host-to-device transfer time", start, end);

	// Setup kernel execution parameters
	// Try different block sizes to find what works best for the hardware
	dim3 blockSize(32, 8); // 32x8 for better memory coalescing
	dim3 gridDim((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

	double totalTime = 0;

	for (size_t step = 0; step < steps; step += stepsPerKernel)
	{
		size_t stepsThisIteration = std::min(stepsPerKernel, steps - step);

		CUDA_CHECK(hipEventRecord(startEvent));

		// Launch kernel without shared memory
		golMultiStepKernel<<<gridDim, blockSize>>>(deviceInput, deviceOutput, deviceTemp, stepsThisIteration);

		CUDA_CHECK(hipEventRecord(endEvent));
		CUDA_CHECK(hipEventSynchronize(endEvent));

		// Check for kernel execution errors
		CUDA_CHECK(hipGetLastError());

		float kernelExecTime = 0;
		CUDA_CHECK(hipEventElapsedTime(&kernelExecTime, startEvent, endEvent));
		logger.log("batch kernel exec time", kernelExecTime);
		totalTime += kernelExecTime;

		// Swap buffers if needed (same logic as OpenCL version)
		if (stepsThisIteration % 2 != 0)
		{
			std::swap(deviceInput, deviceOutput);
		}
	}

	logger.log("total kernel exec time", totalTime);

	// Transfer data back to host
	start = std::chrono::steady_clock::now();

	CUDA_CHECK(hipEventRecord(startEvent));
	CUDA_CHECK(hipMemcpy(hostPinnedOutput, deviceInput, gridSize * sizeof(unsigned char), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipEventRecord(transferEvent));
	CUDA_CHECK(hipEventSynchronize(transferEvent));

	float transferBackTime = 0;
	CUDA_CHECK(hipEventElapsedTime(&transferBackTime, startEvent, transferEvent));
	logger.log("device-to-host transfer time", transferBackTime);

	// Copy the result to output grid
	std::memcpy(outputGrid.data(), hostPinnedOutput, gridSize * sizeof(unsigned char));

	end = std::chrono::steady_clock::now();
	logger.chronoLog("total device-to-host transfer time", start, end);

	// Cleanup
	CUDA_CHECK(hipEventDestroy(transferEvent));
	CUDA_CHECK(hipEventDestroy(startEvent));
	CUDA_CHECK(hipEventDestroy(endEvent));
	CUDA_CHECK(hipHostFree(hostPinnedInput));
	CUDA_CHECK(hipHostFree(hostPinnedOutput));
	CUDA_CHECK(hipFree(deviceInput));
	CUDA_CHECK(hipFree(deviceOutput));
	CUDA_CHECK(hipFree(deviceTemp));
}

int main(int argc, char *argv[])
{
	if (argc == 5)
	{
		const std::string inputFileName = argv[1];
		const std::string outputFileName = argv[2];
		const size_t gameSteps = std::stoll(argv[3]);
		const std::string logFileName = argv[4];

		BenchmarkLogger logger(logFileName, "CUDA");

		auto start = std::chrono::steady_clock::now();

		size_t width;
		size_t height;
		std::vector<unsigned char> grid = loadGridFromFile(inputFileName, width, height);

		auto end = std::chrono::steady_clock::now();

		logger.chronoLog("grid load time", start, end);

		std::vector<unsigned char> outputGrid;

		auto cudaInitStart = std::chrono::steady_clock::now();

		CUDA_CHECK(hipSetDevice(0));

		auto cudaInitEnd = std::chrono::steady_clock::now();

		logger.chronoLog("cuda init time", cudaInitStart, cudaInitEnd);

		unsigned long long w = static_cast<unsigned long long>(width);
		unsigned long long h = static_cast<unsigned long long>(height);

		std::cout << "Processing a " << width << "x" << height << " grid with " << gameSteps << " steps\n";

		auto GoLStart = std::chrono::steady_clock::now();

		GameOfLifeStep(grid, outputGrid, w, h, gameSteps, logger);

		auto GoLEnd = std::chrono::steady_clock::now();

		logger.chronoLog("total game of life time", GoLStart, GoLEnd);

		auto writeGridToFileStart = std::chrono::steady_clock::now();

		writeGridToFile(outputGrid, width, height, outputFileName);

		auto writeGridToFileEnd = std::chrono::steady_clock::now();

		logger.chronoLog("write output grid to file time", writeGridToFileStart, writeGridToFileEnd);
	}
	else
	{
		std::cout << "Correct program usage:\n"
				  << "\t\t" << argv[0] << " <grid file path> <output grid file path> <game steps> <log file path>\n";
	}
	return 0;
}
