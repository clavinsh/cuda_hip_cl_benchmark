#include "hip/hip_runtime.h"
// Game Of Life implementācija CUDA vidē

#include "benchmarkLogger.h"
#include <assert.h>
#include <cassert>
#include <chrono>
#include <cstdint>
#include <cstring>
#include <cuda/std/cstdint> // analogs C/C++ <cstdint>, bet nodrošina fiksētus datu tipu lielumus uz device
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <utility>
#include <vector>

// macro priekš katra cuda API izsaukuma rezultāta pārbaudes
// ņemts no
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CUDA_CHECK(ans)                                                                                                \
	{                                                                                                                  \
		gpuAssert((ans), __FILE__, __LINE__);                                                                          \
	}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

// izveido flat grid masīvu, automātiski nosakot width, height
// met ārā kļūdas ja nav atbilstošu simbolu (1, 0) vai ja kāda rindiņa nesatur tādu pašu simbolu skaitu kā pirmā
std::vector<unsigned char> loadGridFromFile(const std::string &fileName, size_t &width, size_t &height)
{
	// ejam uz faila beigām uzreiz ar 'ate', lai noteiktu faila izmēru, pēc tam iesim uz sākumu
	std::ifstream file(fileName, std::ios::ate | std::ios::binary);

	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t fileSize = file.tellg();
	file.seekg(0, std::ios::beg);

	std::vector<char> buffer(fileSize);
	file.read(buffer.data(), fileSize);
	file.close();

	std::vector<unsigned char> grid;
	grid.reserve(fileSize);

	size_t lineStartPos = 0;

	width = 0;
	height = 0;

	for (size_t i = 0; i <= buffer.size(); ++i)
	{
		if (i == buffer.size() || buffer[i] == '\n')
		{
			size_t lineLen = i - lineStartPos;

			if (lineLen == 0)
			{
				lineStartPos = i + 1;
				continue;
			}

			if (width == 0)
			{
				width = lineLen;
			}
			else if (lineLen != width)
			{
				throw std::runtime_error("Invalid line length at line idx: " + std::to_string(height));
			}

			for (size_t j = 0; j < width; ++j)
			{
				unsigned char val = static_cast<unsigned char>(buffer[lineStartPos + j] - '0');
				grid.push_back(val);
			}

			height++;
			lineStartPos = i + 1;
		}
	}

	return grid;
}

void writeGridToFile(std::vector<unsigned char> &grid, size_t width, size_t height, std::string fileName)
{
	std::ofstream file(fileName, std::ios::out | std::ios::binary);
	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t totalSize = (width + 1) * height; // +1, jo rindas beigās ir \n

	std::vector<char> buffer(totalSize);

	for (size_t h = 0; h < height; h++)
	{
		size_t lineStart = h * (width + 1);
		size_t gridRowStart = h * width;

		for (size_t w = 0; w < width; w++)
		{
			buffer[lineStart + w] = '0' + grid[gridRowStart + w];
		}

		buffer[lineStart + width] = '\n';
	}

	file.write(buffer.data(), totalSize);

	file.close();
}
__global__ void gol_step(const unsigned char *input, unsigned char *output, unsigned long long width,
						 unsigned long long height)
{
	// Calculate global position
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if within bounds
	if (x >= width || y >= height)
		return;

	// Calculate flat index
	size_t idx = y * width + x;

	// Count neighbors
	int neighbors = 0;

	// Top row
	if (y > 0)
	{
		// Top-left
		if (x > 0 && input[(y - 1) * width + (x - 1)] == 1)
			neighbors++;
		// Top-center
		if (input[(y - 1) * width + x] == 1)
			neighbors++;
		// Top-right
		if (x + 1 < width && input[(y - 1) * width + (x + 1)] == 1)
			neighbors++;
	}

	// Middle row
	// Left
	if (x > 0 && input[y * width + (x - 1)] == 1)
		neighbors++;
	// Right
	if (x + 1 < width && input[y * width + (x + 1)] == 1)
		neighbors++;

	// Bottom row
	if (y + 1 < height)
	{
		// Bottom-left
		if (x > 0 && input[(y + 1) * width + (x - 1)] == 1)
			neighbors++;
		// Bottom-center
		if (input[(y + 1) * width + x] == 1)
			neighbors++;
		// Bottom-right
		if (x + 1 < width && input[(y + 1) * width + (x + 1)] == 1)
			neighbors++;
	}

	// Apply Conway's Game of Life rules
	if (input[idx] == 1)
	{
		// Cell is alive
		output[idx] = (neighbors == 2 || neighbors == 3) ? 1 : 0;
	}
	else
	{
		// Cell is dead
		output[idx] = (neighbors == 3) ? 1 : 0;
	}
}

void GameOfLifeStep(std::vector<unsigned char> &grid, std::vector<unsigned char> &outputGrid, unsigned long long width,
					unsigned long long height, size_t steps, BenchmarkLogger &logger)
{
	size_t gridSize = width * height;
	outputGrid.resize(gridSize);

	// Create CUDA events for timing
	hipEvent_t start_event, stop_event, kernel_start, kernel_stop;
	CUDA_CHECK(hipEventCreate(&start_event));
	CUDA_CHECK(hipEventCreate(&stop_event));
	CUDA_CHECK(hipEventCreate(&kernel_start));
	CUDA_CHECK(hipEventCreate(&kernel_stop));
	float milliseconds = 0;

	// Create CUDA streams for overlapping operations
	hipStream_t stream1, stream2;
	CUDA_CHECK(hipStreamCreate(&stream1));
	CUDA_CHECK(hipStreamCreate(&stream2));

	// Start timing memory allocation
	auto start = std::chrono::steady_clock::now();

	// Use pinned memory for faster transfers
	unsigned char *h_pinnedInput = nullptr;
	unsigned char *h_pinnedOutput = nullptr;
	CUDA_CHECK(hipHostMalloc(&h_pinnedInput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipHostMalloc(&h_pinnedOutput, gridSize * sizeof(unsigned char)));

	// Copy data to pinned memory
	std::memcpy(h_pinnedInput, grid.data(), gridSize * sizeof(unsigned char));

	// Allocate device memory
	unsigned char *d_input = nullptr;
	unsigned char *d_output = nullptr;
	CUDA_CHECK(hipMalloc(&d_input, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipMalloc(&d_output, gridSize * sizeof(unsigned char)));

	auto end = std::chrono::steady_clock::now();
	logger.chronoLog("buffer creation time", start, end);

	// Determine block size
	dim3 blockSize(16, 16); // Default block size
	dim3 gridDim((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

	// Record transfer start time
	CUDA_CHECK(hipEventRecord(start_event, stream1));

	// Asynchronously copy input data to device
	CUDA_CHECK(
		hipMemcpyAsync(d_input, h_pinnedInput, gridSize * sizeof(unsigned char), hipMemcpyHostToDevice, stream1));

	// Record transfer end time
	CUDA_CHECK(hipEventRecord(stop_event, stream1));
	CUDA_CHECK(hipEventSynchronize(stop_event));

	CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));
	logger.log("host-to-device transfer time", milliseconds);

	// Record kernel execution start time
	CUDA_CHECK(hipEventRecord(kernel_start, stream1));

	// Process steps using stream1
	for (size_t step = 0; step < steps; step++)
	{
		// Launch kernel for one step in stream1
		gol_step<<<gridDim, blockSize, 0, stream1>>>(d_input, d_output, width, height);

		// Swap buffers for next iteration
		std::swap(d_input, d_output);
	}

	// Make sure the last iteration's result is in d_input
	// (because we swap after each step)
	if (steps % 2 == 1)
	{
		std::swap(d_input, d_output);
	}

	// Record kernel execution end time
	CUDA_CHECK(hipEventRecord(kernel_stop, stream1));
	CUDA_CHECK(hipEventSynchronize(kernel_stop));
	CUDA_CHECK(hipGetLastError()); // Check for any errors in kernel launch

	CUDA_CHECK(hipEventElapsedTime(&milliseconds, kernel_start, kernel_stop));
	logger.log("total kernel exec time", milliseconds);

	// Start timing transfer back to host
	start = std::chrono::steady_clock::now();

	// Record device-to-host transfer start
	CUDA_CHECK(hipEventRecord(start_event, stream2));

	// Asynchronously copy results back to pinned memory using stream2
	// This could potentially overlap with any remaining work in stream1
	CUDA_CHECK(
		hipMemcpyAsync(h_pinnedOutput, d_input, gridSize * sizeof(unsigned char), hipMemcpyDeviceToHost, stream2));

	// Record device-to-host transfer end
	CUDA_CHECK(hipEventRecord(stop_event, stream2));
	CUDA_CHECK(hipEventSynchronize(stop_event));

	CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));
	logger.log("device-to-host transfer time", milliseconds);

	// Copy from pinned memory to final output buffer
	std::memcpy(outputGrid.data(), h_pinnedOutput, gridSize * sizeof(unsigned char));

	end = std::chrono::steady_clock::now();
	logger.chronoLog("total device-to-host transfer time", start, end);

	// Clean up resources
	CUDA_CHECK(hipHostFree(h_pinnedInput));
	CUDA_CHECK(hipHostFree(h_pinnedOutput));
	CUDA_CHECK(hipFree(d_input));
	CUDA_CHECK(hipFree(d_output));
	CUDA_CHECK(hipEventDestroy(start_event));
	CUDA_CHECK(hipEventDestroy(stop_event));
	CUDA_CHECK(hipEventDestroy(kernel_start));
	CUDA_CHECK(hipEventDestroy(kernel_stop));
	CUDA_CHECK(hipStreamDestroy(stream1));
	CUDA_CHECK(hipStreamDestroy(stream2));
}

int main(int argc, char *argv[])
{
	if (argc == 5)
	{
		const std::string inputFileName = argv[1];
		const std::string outputFileName = argv[2];
		const size_t gameSteps = std::stoll(argv[3]);
		const std::string logFileName = argv[4];

		BenchmarkLogger logger(logFileName, "CUDA");

		auto start = std::chrono::steady_clock::now();

		size_t width;
		size_t height;
		std::vector<unsigned char> grid = loadGridFromFile(inputFileName, width, height);

		auto end = std::chrono::steady_clock::now();

		logger.chronoLog("grid load time", start, end);

		std::vector<unsigned char> outputGrid;

		auto cudaInitStart = std::chrono::steady_clock::now();

		CUDA_CHECK(hipSetDevice(0));

		auto cudaInitEnd = std::chrono::steady_clock::now();

		logger.chronoLog("cuda init time", cudaInitStart, cudaInitEnd);

		unsigned long long w = static_cast<unsigned long long>(width);
		unsigned long long h = static_cast<unsigned long long>(height);

		std::cout << "Processing a " << width << "x" << height << " grid with " << gameSteps << " steps\n";

		auto GoLStart = std::chrono::steady_clock::now();

		GameOfLifeStep(grid, outputGrid, w, h, gameSteps, logger);

		auto GoLEnd = std::chrono::steady_clock::now();

		logger.chronoLog("total game of life time", GoLStart, GoLEnd);

		auto writeGridToFileStart = std::chrono::steady_clock::now();

		writeGridToFile(outputGrid, width, height, outputFileName);

		auto writeGridToFileEnd = std::chrono::steady_clock::now();

		logger.chronoLog("write output grid to file time", writeGridToFileStart, writeGridToFileEnd);
	}
	else
	{
		std::cout << "Correct program usage:\n"
				  << "\t\t" << argv[0] << " <grid file path> <output grid file path> <game steps> <log file path>\n";
	}
	return 0;
}
