#include "hip/hip_runtime.h"
// Game Of Life implementācija CUDA vidē

#include "benchmarkLogger.h"
#include <assert.h>
#include <cassert>
#include <chrono>
#include <cstring>
#include <cuda/std/cstdint> // analogs C/C++ <cstdint>, bet nodrošina fiksētus datu tipu lielumus uz device
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <utility>
#include <vector>

// macro priekš katra cuda API izsaukuma rezultāta pārbaudes
// ņemts no
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CUDA_CHECK(ans)                                                                                                \
	{                                                                                                                  \
		gpuAssert((ans), __FILE__, __LINE__);                                                                          \
	}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

// izveido flat grid masīvu, automātiski nosakot width, height
// met ārā kļūdas ja nav atbilstošu simbolu (1, 0) vai ja kāda rindiņa nesatur tādu pašu simbolu skaitu kā pirmā
std::vector<unsigned char> loadGridFromFile(const std::string &fileName, size_t &width, size_t &height)
{
	// ejam uz faila beigām uzreiz ar 'ate', lai noteiktu faila izmēru, pēc tam iesim uz sākumu
	std::ifstream file(fileName, std::ios::ate | std::ios::binary);

	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t fileSize = file.tellg();
	file.seekg(0, std::ios::beg);

	std::vector<char> buffer(fileSize);
	file.read(buffer.data(), fileSize);
	file.close();

	std::vector<unsigned char> grid;
	grid.reserve(fileSize);

	size_t lineStartPos = 0;

	width = 0;
	height = 0;

	for (size_t i = 0; i <= buffer.size(); ++i)
	{
		if (i == buffer.size() || buffer[i] == '\n')
		{
			size_t lineLen = i - lineStartPos;

			if (lineLen == 0)
			{
				lineStartPos = i + 1;
				continue;
			}

			if (width == 0)
			{
				width = lineLen;
			}
			else if (lineLen != width)
			{
				throw std::runtime_error("Invalid line length at line idx: " + std::to_string(height));
			}

			for (size_t j = 0; j < width; ++j)
			{
				unsigned char val = static_cast<unsigned char>(buffer[lineStartPos + j] - '0');
				grid.push_back(val);
			}

			height++;
			lineStartPos = i + 1;
		}
	}

	return grid;
}

void writeGridToFile(std::vector<unsigned char> &grid, size_t width, size_t height, std::string fileName)
{
	std::ofstream file(fileName, std::ios::out | std::ios::binary);
	if (!file.is_open())
	{
		throw std::runtime_error("Failed to open file: " + fileName);
	}

	const size_t totalSize = (width + 1) * height; // +1, jo rindas beigās ir \n

	std::vector<char> buffer(totalSize);

	for (size_t h = 0; h < height; h++)
	{
		size_t lineStart = h * (width + 1);
		size_t gridRowStart = h * width;

		for (size_t w = 0; w < width; w++)
		{
			buffer[lineStart + w] = '0' + grid[gridRowStart + w];
		}

		buffer[lineStart + width] = '\n';
	}

	file.write(buffer.data(), totalSize);

	file.close();
}

__constant__ size_t d_width;
__constant__ size_t d_height;

inline __device__ int neighborCount(int x, int y, const unsigned char *grid)
{
	int neighbors = 0;

	if (y > 0)
	{
		if (x > 0)
			neighbors += grid[(y - 1) * d_width + (x - 1)];

		neighbors += grid[(y - 1) * d_width + x];

		if (x < d_width - 1)
			neighbors += grid[(y - 1) * d_width + (x + 1)];
	}

	if (x > 0)
		neighbors += grid[y * d_width + (x - 1)];

	if (x < d_width - 1)
		neighbors += grid[y * d_width + (x + 1)];

	if (y < d_height - 1)
	{
		if (x > 0)
			neighbors += grid[(y + 1) * d_width + (x - 1)];

		neighbors += grid[(y + 1) * d_width + x];

		if (x < d_width - 1)
			neighbors += grid[(y + 1) * d_width + (x + 1)];
	}

	return neighbors;
}

__global__ void golMultiStepKernel(unsigned char *input, unsigned char *output)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= d_width || y >= d_height)
		return;

	const size_t flatIdx = y * d_width + x;

	int neighbors = neighborCount(x, y, input);

	unsigned char cell = 0;
	if (input[flatIdx] == 1)
	{
		if (neighbors == 2 || neighbors == 3)
			cell = 1;
	}
	else
	{
		if (neighbors == 3)
			cell = 1;
	}

	output[flatIdx] = cell;
}

void GameOfLifeStep(std::vector<unsigned char> &grid, std::vector<unsigned char> &outputGrid, size_t width,
					size_t height, size_t steps, BenchmarkLogger &logger)
{

	size_t gridSize = width * height;
	outputGrid.resize(gridSize);

	auto start = std::chrono::steady_clock::now();

	unsigned char *hostPinnedInput = nullptr;
	unsigned char *hostPinnedOutput = nullptr;
	CUDA_CHECK(hipHostMalloc(&hostPinnedInput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipHostMalloc(&hostPinnedOutput, gridSize * sizeof(unsigned char)));

	std::memcpy(hostPinnedInput, grid.data(), gridSize * sizeof(unsigned char));

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(size_t)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(size_t)));

	unsigned char *deviceInput = nullptr;
	unsigned char *deviceOutput = nullptr;
	CUDA_CHECK(hipMalloc(&deviceInput, gridSize * sizeof(unsigned char)));
	CUDA_CHECK(hipMalloc(&deviceOutput, gridSize * sizeof(unsigned char)));

	auto end = std::chrono::steady_clock::now();

	logger.chronoLog("buffer creation time", start, end);

	start = std::chrono::steady_clock::now();

	hipEvent_t transferEvent, startEvent, endEvent;
	CUDA_CHECK(hipEventCreate(&transferEvent));
	CUDA_CHECK(hipEventCreate(&startEvent));
	CUDA_CHECK(hipEventCreate(&endEvent));

	CUDA_CHECK(hipEventRecord(startEvent));
	CUDA_CHECK(hipMemcpy(deviceInput, hostPinnedInput, gridSize * sizeof(unsigned char), hipMemcpyHostToDevice));
	CUDA_CHECK(hipEventRecord(transferEvent));
	CUDA_CHECK(hipEventSynchronize(transferEvent));

	float transferTime = 0;
	CUDA_CHECK(hipEventElapsedTime(&transferTime, startEvent, transferEvent));
	logger.log("host-to-device transfer time", transferTime);

	end = std::chrono::steady_clock::now();
	logger.chronoLog("total host-to-device transfer time", start, end);

	// lokālais bloka izmērs, šis likās diezgan ok
	dim3 blockSize(32, 8);
	dim3 gridDim((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

	double totalTime = 0;

	unsigned char *currentInput = deviceInput;
	unsigned char *currentOutput = deviceOutput;

	for (size_t step = 0; step < steps; step++)
	{

		CUDA_CHECK(hipEventRecord(startEvent));

		golMultiStepKernel<<<gridDim, blockSize>>>(currentInput, currentOutput);

		CUDA_CHECK(hipEventRecord(endEvent));
		CUDA_CHECK(hipEventSynchronize(endEvent));

		CUDA_CHECK(hipGetLastError());

		float kernelExecTime = 0;
		CUDA_CHECK(hipEventElapsedTime(&kernelExecTime, startEvent, endEvent));
		logger.log("kernel exec time", kernelExecTime);
		totalTime += kernelExecTime;

		std::swap(currentInput, currentOutput);
	}

	logger.log("total kernel exec time", totalTime);

	start = std::chrono::steady_clock::now();

	CUDA_CHECK(hipEventRecord(startEvent));
	// ņemot vērā pēdējo std::swap ar buferiem, pēdējā izeja atrodas input bufeŗi
	CUDA_CHECK(hipMemcpy(hostPinnedOutput, currentInput, gridSize * sizeof(unsigned char), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipEventRecord(transferEvent));
	CUDA_CHECK(hipEventSynchronize(transferEvent));

	float transferBackTime = 0;
	CUDA_CHECK(hipEventElapsedTime(&transferBackTime, startEvent, transferEvent));

	logger.log("device-to-host transfer time", transferBackTime);

	std::memcpy(outputGrid.data(), hostPinnedOutput, gridSize * sizeof(unsigned char));

	end = std::chrono::steady_clock::now();
	logger.chronoLog("total device-to-host transfer time", start, end);

	CUDA_CHECK(hipEventDestroy(transferEvent));
	CUDA_CHECK(hipEventDestroy(startEvent));
	CUDA_CHECK(hipEventDestroy(endEvent));
	CUDA_CHECK(hipHostFree(hostPinnedInput));
	CUDA_CHECK(hipHostFree(hostPinnedOutput));
	CUDA_CHECK(hipFree(deviceInput));
	CUDA_CHECK(hipFree(deviceOutput));
}

int main(int argc, char *argv[])
{
	if (argc == 5)
	{
		const std::string inputFileName = argv[1];
		const std::string outputFileName = argv[2];
		const size_t gameSteps = std::stoll(argv[3]);
		const std::string logFileName = argv[4];

		BenchmarkLogger logger(logFileName, "CUDA");

		auto start = std::chrono::steady_clock::now();

		size_t width;
		size_t height;
		std::vector<unsigned char> grid = loadGridFromFile(inputFileName, width, height);

		auto end = std::chrono::steady_clock::now();

		logger.chronoLog("grid load time", start, end);

		std::vector<unsigned char> outputGrid;

		auto cudaInitStart = std::chrono::steady_clock::now();

		CUDA_CHECK(hipSetDevice(0));

		auto cudaInitEnd = std::chrono::steady_clock::now();

		logger.chronoLog("cuda init time", cudaInitStart, cudaInitEnd);

		unsigned long long w = static_cast<unsigned long long>(width);
		unsigned long long h = static_cast<unsigned long long>(height);

		std::cout << "Processing a " << width << "x" << height << " grid with " << gameSteps << " steps\n";

		auto GoLStart = std::chrono::steady_clock::now();

		GameOfLifeStep(grid, outputGrid, w, h, gameSteps, logger);

		auto GoLEnd = std::chrono::steady_clock::now();

		logger.chronoLog("total game of life time", GoLStart, GoLEnd);

		auto writeGridToFileStart = std::chrono::steady_clock::now();

		writeGridToFile(outputGrid, width, height, outputFileName);

		auto writeGridToFileEnd = std::chrono::steady_clock::now();

		logger.chronoLog("write output grid to file time", writeGridToFileStart, writeGridToFileEnd);
	}
	else
	{
		std::cout << "Correct program usage:\n"
				  << "\t\t" << argv[0] << " <grid file path> <output grid file path> <game steps> <log file path>\n";
	}
	return 0;
}
